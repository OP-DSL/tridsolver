#define CATCH_CONFIG_NOSTDOUT
#include "catch.hpp"
#include "catch_mpi_outputs.hpp"
#include "cuda_utils.hpp"
#include "catch_utils.hpp"
#include "cuda_mpi_wrappers.hpp"

#include "trid_cuda_mpi_pcr.hpp"

#include <trid_common.h>
#include <trid_cuda.h>
#include <trid_mpi_cuda.hpp>


#include <mpi.h>

#include <chrono>
#include <cstdlib>
#include <functional>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <sstream>
#include <thread>

template <typename Float, int INC, MpiSolverParams::MPICommStrategy strategy>
void test_solver_from_file(const std::string &file_name) {
  // The dimension of the MPI decomposition is the same as solve_dim
  MeshLoader<Float> mesh(file_name);

  int num_proc, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &num_proc);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  // Create rectangular grid
  std::vector<int> mpi_dims(mesh.dims().size(), 0),
      periods(mesh.dims().size(), 0);
  mpi_dims[mesh.solve_dim()] = num_proc;
  MPI_Dims_create(num_proc, mesh.dims().size(), mpi_dims.data());

  // Create communicator for grid
  MPI_Comm cart_comm;
  MPI_Cart_create(MPI_COMM_WORLD, mesh.dims().size(), mpi_dims.data(),
                  periods.data(), 0, &cart_comm);

  MpiSolverParams params(cart_comm, mesh.dims().size(), mpi_dims.data(), 256,
                         strategy);

  // The size of the local domain.
  std::vector<int> local_sizes(mesh.dims().size());
  // The starting indices of the local domain in each dimension.
  std::vector<int> domain_offsets(mesh.dims().size());
  // The strides in the mesh for each dimension.
  std::vector<int> global_strides(mesh.dims().size());
  int domain_size = 1;
  for (size_t i = 0; i < local_sizes.size(); ++i) {
    const int global_dim = mesh.dims()[i];
    domain_offsets[i]    = params.mpi_coords[i] * (global_dim / mpi_dims[i]);
    local_sizes[i]       = params.mpi_coords[i] == mpi_dims[i] - 1
                               ? global_dim - domain_offsets[i]
                               : global_dim / mpi_dims[i];
    global_strides[i] = i == 0 ? 1 : global_strides[i - 1] * mesh.dims()[i - 1];
    domain_size *= local_sizes[i];
  }

  // Simulate distributed environment: only load our data
  AlignedArray<Float, 1> a(domain_size), b(domain_size), c(domain_size),
      u(domain_size), d(domain_size);
  copy_strided(mesh.a(), a, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);
  copy_strided(mesh.b(), b, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);
  copy_strided(mesh.c(), c, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);
  copy_strided(mesh.d(), d, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);
  copy_strided(mesh.u(), u, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);

  GPUMesh<Float> local_device_mesh(a, b, c, d, local_sizes);

  // Solve the equations
  std::vector<Float> host_init(domain_size, 0);
  DeviceArray<Float> u_d(host_init.data(), domain_size);
  tridmtsvStridedBatchMPIWrapper<Float, INC>(
      params, local_device_mesh.a().data(), local_device_mesh.b().data(),
      local_device_mesh.c().data(), local_device_mesh.d().data(), u_d.data(),
      mesh.dims().size(), mesh.solve_dim(), local_sizes.data(),
      local_sizes.data());

  if (!INC) {
    hipMemcpy(d.data(), local_device_mesh.d().data(),
               sizeof(Float) * domain_size, hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(d.data(), u_d.data(), sizeof(Float) * domain_size,
               hipMemcpyDeviceToHost);
  }
  // Check result
  require_allclose(u, d, domain_size, 1);
}

template <typename Float, int INC, MpiSolverParams::MPICommStrategy strategy>
void test_solver_from_file_padded(const std::string &file_name) {
  // The dimension of the MPI decomposition is the same as solve_dim
  MeshLoader<Float> mesh(file_name);

  int num_proc, rank;
  MPI_Comm_size(MPI_COMM_WORLD, &num_proc);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  // Create rectangular grid
  std::vector<int> mpi_dims(mesh.dims().size(), 0),
      periods(mesh.dims().size(), 0);
  mpi_dims[mesh.solve_dim()] = num_proc;
  MPI_Dims_create(num_proc, mesh.dims().size(), mpi_dims.data());

  // Create communicator for grid
  MPI_Comm cart_comm;
  MPI_Cart_create(MPI_COMM_WORLD, mesh.dims().size(), mpi_dims.data(),
                  periods.data(), 0, &cart_comm);

  MpiSolverParams params(cart_comm, mesh.dims().size(), mpi_dims.data(), 256,
                         strategy);

  // The size of the local domain.
  std::vector<int> local_sizes(mesh.dims().size());
  // The starting indices of the local domain in each dimension.
  std::vector<int> domain_offsets(mesh.dims().size());
  // The strides in the mesh for each dimension.
  std::vector<int> global_strides(mesh.dims().size());
  int domain_size = 1;
  for (size_t i = 0; i < local_sizes.size(); ++i) {
    const int global_dim = mesh.dims()[i];
    domain_offsets[i]    = params.mpi_coords[i] * (global_dim / mpi_dims[i]);
    local_sizes[i]       = params.mpi_coords[i] == mpi_dims[i] - 1
                               ? global_dim - domain_offsets[i]
                               : global_dim / mpi_dims[i];
    global_strides[i] = i == 0 ? 1 : global_strides[i - 1] * mesh.dims()[i - 1];
    domain_size *= local_sizes[i];
  }

  // Simulate distributed environment: only load our data
  AlignedArray<Float, 1> a(domain_size), b(domain_size), c(domain_size),
      u(domain_size), d(domain_size);
  copy_strided(mesh.a(), a, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);
  copy_strided(mesh.b(), b, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);
  copy_strided(mesh.c(), c, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);
  copy_strided(mesh.d(), d, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);
  copy_strided(mesh.u(), u, local_sizes, domain_offsets, global_strides,
               local_sizes.size() - 1);

  std::vector<int> padded_dims = local_sizes;
  int padded_size              = 1;
  for (int i = 0; i < padded_dims.size(); i++) {
    padded_dims[i] += 2;
    padded_size *= padded_dims[i];
  }

  std::vector<Float> a_p(padded_size);
  std::vector<Float> b_p(padded_size);
  std::vector<Float> c_p(padded_size);
  std::vector<Float> d_p(padded_size);
  std::vector<Float> u_p(padded_size);
  std::vector<Float> u_zero(padded_size, 0);

  copy_to_padded_array(a, a_p, local_sizes);
  copy_to_padded_array(b, b_p, local_sizes);
  copy_to_padded_array(c, c_p, local_sizes);
  copy_to_padded_array(d, d_p, local_sizes);
  copy_to_padded_array(u, u_p, local_sizes);

  int offset_to_first_element =
      padded_dims[1] * padded_dims[0] + padded_dims[0] + 1;

  Float *a_d, *b_d, *c_d, *d_d, *u_d;
  hipMalloc((void **)&a_d, padded_size * sizeof(Float));
  hipMalloc((void **)&b_d, padded_size * sizeof(Float));
  hipMalloc((void **)&c_d, padded_size * sizeof(Float));
  hipMalloc((void **)&d_d, padded_size * sizeof(Float));
  hipMalloc((void **)&u_d, padded_size * sizeof(Float));

  hipMemcpy(a_d, a_p.data(), a_p.size() * sizeof(Float),
             hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_p.data(), b_p.size() * sizeof(Float),
             hipMemcpyHostToDevice);
  hipMemcpy(c_d, c_p.data(), c_p.size() * sizeof(Float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_d, d_p.data(), d_p.size() * sizeof(Float),
             hipMemcpyHostToDevice);
  hipMemcpy(u_d, u_zero.data(), d_p.size() * sizeof(Float),
             hipMemcpyHostToDevice);

  tridmtsvStridedBatchMPIWrapper<Float, INC>(
      params, a_d + offset_to_first_element, b_d + offset_to_first_element,
      c_d + offset_to_first_element, d_d + offset_to_first_element,
      u_d + offset_to_first_element, mesh.dims().size(), mesh.solve_dim(),
      local_sizes.data(), padded_dims.data(), offset_to_first_element);

  if (!INC) {
    hipMemcpy(d_p.data(), d_d, sizeof(Float) * d_p.size(),
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(d_p.data(), u_d, sizeof(Float) * d_p.size(),
               hipMemcpyDeviceToHost);
  }
  // Check result
  require_allclose_padded(u_p, d_p);
}

template <typename Float>
void test_PCR_on_reduced(const std::string &file_name) {
  // reduced system:
  //  b is 1 everywhere
  //  consider each 2 element as a result of the forward run for separate mpi
  //  nodes
  //  one input from merged a c and d arrays
  //    layout: [aa0 aa1 cc0 cc1 dd0 dd1 (for each system) aa1 aa2 bb1 bb2 ...]
  //    size: 6 * sys_n * mpi_process_num
  //  one output with the 2 d values per system based on mpi coord
  //    layout: [d_[2*mpi_proc_id] d_[2*mpi_proc_id + 1] ...(for every system)]
  //    size: 2 * sys_n

  // AlignedArray<double, 1> aa(mesh.a()), cc(mesh.c()), dd(mesh.d());
  MeshLoader<Float> mesh(file_name);
  const int reduced_sys_len = mesh.dims()[mesh.solve_dim()];
  const int num_mpi_procs   = reduced_sys_len / 2;
  const int mpi_coord       = num_mpi_procs / 2;
  const int sys_n =
      std::accumulate(mesh.dims().begin() + mesh.solve_dim() + 1,
                      mesh.dims().end(), 1, std::multiplies<int>{});
  // buffer holding the 3 arrays (a, c, d) merged:
  AlignedArray<Float, 1> buffer(sys_n * reduced_sys_len * 3);
  for (int mpi_coord = 0; mpi_coord < num_mpi_procs; ++mpi_coord) {
    for (int sys_idx = 0; sys_idx < sys_n; ++sys_idx) {
      buffer.push_back(mesh.a()[sys_idx * reduced_sys_len + 2 * mpi_coord]);
      buffer.push_back(mesh.a()[sys_idx * reduced_sys_len + 2 * mpi_coord + 1]);
      buffer.push_back(mesh.c()[sys_idx * reduced_sys_len + 2 * mpi_coord]);
      buffer.push_back(mesh.c()[sys_idx * reduced_sys_len + 2 * mpi_coord + 1]);
      buffer.push_back(mesh.d()[sys_idx * reduced_sys_len + 2 * mpi_coord]);
      buffer.push_back(mesh.d()[sys_idx * reduced_sys_len + 2 * mpi_coord + 1]);
    }
  }
  DeviceArray<Float> buffer_d(buffer);
  DeviceArray<Float> result_d(2 * sys_n);

  pcr_on_reduced_batched<Float>(buffer_d.data(), result_d.data(), sys_n,
                                mpi_coord, reduced_sys_len);

  AlignedArray<Float, 1> result(2 * sys_n);
  result.resize(2 * sys_n);
  hipMemcpy(result.data(), result_d.data(), sizeof(Float) * 2 * sys_n,
             hipMemcpyDeviceToHost);
  // BATCHING reduced calls
  const int batch_size  = 32;
  const int num_batches = 1 + (sys_n - 1) / batch_size;
  AlignedArray<Float, 1> batched_buffer(sys_n * reduced_sys_len * 3);
  for (int bidx = 0; bidx < num_batches; ++bidx) {
    int batch_start = bidx * batch_size;
    int bsize = bidx == num_batches - 1 ? sys_n - batch_start : batch_size;
    // Solve the reduced system
    for (int mpi_coord = 0; mpi_coord < num_mpi_procs; ++mpi_coord) {
      for (int sys_idx = batch_start; sys_idx < batch_start + bsize;
           ++sys_idx) {
        batched_buffer.push_back(
            mesh.a()[sys_idx * reduced_sys_len + 2 * mpi_coord]);
        batched_buffer.push_back(
            mesh.a()[sys_idx * reduced_sys_len + 2 * mpi_coord + 1]);
        batched_buffer.push_back(
            mesh.c()[sys_idx * reduced_sys_len + 2 * mpi_coord]);
        batched_buffer.push_back(
            mesh.c()[sys_idx * reduced_sys_len + 2 * mpi_coord + 1]);
        batched_buffer.push_back(
            mesh.d()[sys_idx * reduced_sys_len + 2 * mpi_coord]);
        batched_buffer.push_back(
            mesh.d()[sys_idx * reduced_sys_len + 2 * mpi_coord + 1]);
      }
    }
  }
  DeviceArray<Float> buffer_batched_d(batched_buffer);
  DeviceArray<Float> result_batched_d(2 * sys_n);
  const int sys_bound_size = 6;
  for (int bidx = 0; bidx < num_batches; ++bidx) {
    int batch_start = bidx * batch_size;
    int bsize = bidx == num_batches - 1 ? sys_n - batch_start : batch_size;
    // Solve the reduced system
    int buf_offset       = sys_bound_size * num_mpi_procs * batch_start;
    int bound_buf_offset = 2 * batch_start;
    pcr_on_reduced_batched<Float>(buffer_batched_d.data() + buf_offset,
                                  result_batched_d.data() + bound_buf_offset,
                                  bsize, mpi_coord, reduced_sys_len);
  }
  AlignedArray<Float, 1> result_batched(2 * sys_n);
  result_batched.resize(2 * sys_n);
  hipMemcpy(result_batched.data(), result_batched_d.data(),
             sizeof(Float) * 2 * sys_n, hipMemcpyDeviceToHost);
  require_allclose(result, result_batched);
}

TEMPLATE_TEST_CASE("PCR on reduced", "[reduced]", double, float) {
  test_PCR_on_reduced<TestType>("files/reduced_test_small");
}

enum ResDest { assign = 0, increment };

#define PARAM_COMBOS                                                           \
  (double, assign, MpiSolverParams::ALLGATHER),                                \
      (double, assign, MpiSolverParams::LATENCY_HIDING_INTERLEAVED),           \
      (double, assign, MpiSolverParams::LATENCY_HIDING_TWO_STEP),              \
      (float, assign, MpiSolverParams::ALLGATHER),                             \
      (float, assign, MpiSolverParams::LATENCY_HIDING_INTERLEAVED),            \
      (float, assign, MpiSolverParams::LATENCY_HIDING_TWO_STEP),               \
      (double, increment, MpiSolverParams::ALLGATHER),                         \
      (double, increment, MpiSolverParams::LATENCY_HIDING_INTERLEAVED),        \
      (double, increment, MpiSolverParams::LATENCY_HIDING_TWO_STEP),           \
      (float, increment, MpiSolverParams::ALLGATHER),                          \
      (float, increment, MpiSolverParams::LATENCY_HIDING_INTERLEAVED),         \
      (float, increment, MpiSolverParams::LATENCY_HIDING_TWO_STEP)

TEMPLATE_TEST_CASE_SIG("cuda solver mpi: solveX", "[solver][solvedim:0]",
                       ((typename TestType, ResDest INC,
                         MpiSolverParams::MPICommStrategy strategy),
                        TestType, INC, strategy),
                       PARAM_COMBOS) {
  SECTION("ndims: 1") {
    test_solver_from_file<TestType, INC, strategy>("files/one_dim_large");
  }
  SECTION("ndims: 2") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/two_dim_large_solve0");
  }
  SECTION("ndims: 3") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/three_dim_large_solve0");
  }
}

TEMPLATE_TEST_CASE_SIG("cuda solver mpi: solveY", "[solver][solvedim:1]",
                       ((typename TestType, ResDest INC,
                         MpiSolverParams::MPICommStrategy strategy),
                        TestType, INC, strategy),
                       PARAM_COMBOS) {
  SECTION("ndims: 2") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/two_dim_large_solve1");
  }
  SECTION("ndims: 3") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/three_dim_large_solve1");
  }
}

TEMPLATE_TEST_CASE_SIG("cuda solver mpi: solveZ", "[solver][solvedim:2]",
                       ((typename TestType, ResDest INC,
                         MpiSolverParams::MPICommStrategy strategy),
                        TestType, INC, strategy),
                       PARAM_COMBOS) {
  SECTION("ndims: 3") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/three_dim_large_solve2");
  }
}

TEMPLATE_TEST_CASE_SIG("cuda: padded", "[padded]",
                       ((typename TestType, ResDest INC,
                         MpiSolverParams::MPICommStrategy strategy),
                        TestType, INC, strategy),
                       PARAM_COMBOS) {
  SECTION("ndims: 3") {
    SECTION("solvedim: 0") {
      test_solver_from_file_padded<TestType, INC, strategy>(
          "files/three_dim_large_solve0");
    }
    SECTION("solvedim: 1") {
      test_solver_from_file_padded<TestType, INC, strategy>(
          "files/three_dim_large_solve1");
    }
    SECTION("solvedim: 2") {
      test_solver_from_file_padded<TestType, INC, strategy>(
          "files/three_dim_large_solve2");
    }
  }
}

#if MAXDIM > 3
TEMPLATE_TEST_CASE_SIG("cuda solver mpi 4D: solveX", "[solver][solvedim:0]",
                       ((typename TestType, ResDest INC,
                         MpiSolverParams::MPICommStrategy strategy),
                        TestType, INC, strategy),
                       PARAM_COMBOS) {
  SECTION("ndims: 4") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/four_dim_large_solve0");
  }
}

TEMPLATE_TEST_CASE_SIG("cuda solver mpi 4D: solveY", "[solver][solvedim:1]",
                       ((typename TestType, ResDest INC,
                         MpiSolverParams::MPICommStrategy strategy),
                        TestType, INC, strategy),
                       PARAM_COMBOS) {
  SECTION("ndims: 4") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/four_dim_large_solve1");
  }
}

TEMPLATE_TEST_CASE_SIG("cuda solver mpi 4D: solveZ", "[solver][solvedim:2]",
                       ((typename TestType, ResDest INC,
                         MpiSolverParams::MPICommStrategy strategy),
                        TestType, INC, strategy),
                       PARAM_COMBOS) {
  SECTION("ndims: 4") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/four_dim_large_solve2");
  }
}

TEMPLATE_TEST_CASE_SIG("cuda solver mpi: solve3", "[solver][solvedim:3]",
                       ((typename TestType, ResDest INC,
                         MpiSolverParams::MPICommStrategy strategy),
                        TestType, INC, strategy),
                       PARAM_COMBOS) {
  SECTION("ndims: 4") {
    test_solver_from_file<TestType, INC, strategy>(
        "files/four_dim_large_solve3");
  }
}
#endif
