#include "catch.hpp"
#include "cuda_utils.hpp"
#include "catch_utils.hpp"

#include <trid_cuda.h>

template <typename Float>
tridStatus_t tridStridedBatchWrapper(const Float *a, const Float *b,
                                     const Float *c, Float *d, Float *u,
                                     int ndim, int solvedim, int *dims,
                                     int *pads);

template <>
tridStatus_t tridStridedBatchWrapper<float>(const float *a, const float *b,
                                            const float *c, float *d, float *u,
                                            int ndim, int solvedim, int *dims,
                                            int *pads) {
  int opts[] = {0, 0, 0};
  return tridSmtsvStridedBatch(a, b, c, d, u, ndim, solvedim, dims, pads, opts,
                               0);
}

template <>
tridStatus_t tridStridedBatchWrapper<double>(const double *a, const double *b,
                                             const double *c, double *d,
                                             double *u, int ndim, int solvedim,
                                             int *dims, int *pads) {
  int opts[] = {0, 0, 0};
  return tridDmtsvStridedBatch(a, b, c, d, u, ndim, solvedim, dims, pads, opts,
                               0);
}

template <typename Float> void test_from_file(const std::string &file_name) {
  MeshLoader<Float> mesh(file_name);
  std::vector<int> dims = mesh.dims(); // Because it isn't const in the lib
  while (dims.size() < 3) {
    dims.push_back(1);
  }
  GPUMesh<Float> device_mesh(mesh);

  const tridStatus_t status =
      tridStridedBatchWrapper<Float>(device_mesh.a().data(), // a
                                     device_mesh.b().data(), // b
                                     device_mesh.c().data(), // c
                                     device_mesh.d().data(), // d
                                     nullptr,                // u
                                     mesh.dims().size(),     // ndim
                                     mesh.solve_dim(),       // solvedim
                                     dims.data(),            // dims
                                     dims.data());           // pads

  CHECK(status == TRID_STATUS_SUCCESS);

  AlignedArray<Float, 1> d(mesh.d());
  hipMemcpy(d.data(), device_mesh.d().data(), d.size() * sizeof(Float),
             hipMemcpyDeviceToHost);
  require_allclose(mesh.u(), d);
}

template <typename Float>
void test_from_file_padded(const std::string &file_name) {
  MeshLoader<Float> mesh(file_name);
  std::vector<int> dims = mesh.dims(); // Because it isn't const in the lib
  while (dims.size() < 3) {
    dims.push_back(1);
  }

  std::vector<int> padded_dims = dims;
  int padded_size              = 1;
  for (int i = 0; i < padded_dims.size(); i++) {
    padded_dims[i] += 2;
    padded_size *= padded_dims[i];
  }

  std::vector<Float> a(padded_size);
  std::vector<Float> b(padded_size);
  std::vector<Float> c(padded_size);
  std::vector<Float> d(padded_size);
  std::vector<Float> u(padded_size);

  copy_to_padded_array(mesh.a(), a, dims);
  copy_to_padded_array(mesh.b(), b, dims);
  copy_to_padded_array(mesh.c(), c, dims);
  copy_to_padded_array(mesh.d(), d, dims);
  copy_to_padded_array(mesh.u(), u, dims);

  Float *a_d, *b_d, *c_d, *d_d;
  hipMalloc((void **)&a_d, padded_size * sizeof(Float));
  hipMalloc((void **)&b_d, padded_size * sizeof(Float));
  hipMalloc((void **)&c_d, padded_size * sizeof(Float));
  hipMalloc((void **)&d_d, padded_size * sizeof(Float));

  hipMemcpy(a_d, a.data(), a.size() * sizeof(Float), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b.data(), b.size() * sizeof(Float), hipMemcpyHostToDevice);
  hipMemcpy(c_d, c.data(), c.size() * sizeof(Float), hipMemcpyHostToDevice);
  hipMemcpy(d_d, d.data(), d.size() * sizeof(Float), hipMemcpyHostToDevice);

  int offset_to_first_element =
      padded_dims[1] * padded_dims[0] + padded_dims[0] + 1;

  const tridStatus_t status =
      tridStridedBatchWrapper<Float>(a_d + offset_to_first_element, // a
                                     b_d + offset_to_first_element, // b
                                     c_d + offset_to_first_element, // c
                                     d_d + offset_to_first_element, // d
                                     nullptr,                       // u
                                     mesh.dims().size(),            // ndim
                                     mesh.solve_dim(),              // solvedim
                                     dims.data(),                   // dims
                                     padded_dims.data());           // pads

  CHECK(status == TRID_STATUS_SUCCESS);

  hipMemcpy(d.data(), d_d, d.size() * sizeof(Float), hipMemcpyDeviceToHost);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  hipFree(d_d);
  require_allclose_padded(u, d);
}

TEMPLATE_TEST_CASE("cuda: solveX", "[solvedim:0]", double, float) {
  SECTION("ndims: 2") {
    test_from_file<TestType>("files/two_dim_large_solve0");
  }
  SECTION("ndims: 3") {
    test_from_file<TestType>("files/three_dim_large_solve0");
  }
}

TEMPLATE_TEST_CASE("cuda: solveY", "[solvedim:1]", double, float) {
  SECTION("ndims: 2") {
    test_from_file<TestType>("files/two_dim_large_solve1");
  }
  SECTION("ndims: 3") {
    test_from_file<TestType>("files/three_dim_large_solve1");
  }
}

TEMPLATE_TEST_CASE("cuda: solveZ", "[solvedim:2]", double, float) {
  SECTION("ndims: 3") {
    SECTION("solvedim: 2") {
      test_from_file<TestType>("files/three_dim_large_solve2");
    }
  }
}

TEMPLATE_TEST_CASE("cuda: padded", "[padded]", double, float) {
  SECTION("ndims: 3") {
    SECTION("solvedim: 0") {
      test_from_file_padded<TestType>("files/three_dim_large_solve0");
    }
    SECTION("solvedim: 1") {
      test_from_file_padded<TestType>("files/three_dim_large_solve1");
    }
    SECTION("solvedim: 2") {
      test_from_file_padded<TestType>("files/three_dim_large_solve2");
    }
  }
}

#if MAXDIM > 3
TEMPLATE_TEST_CASE("cuda 4D: solveX", "[solvedim:0]", double, float) {
  SECTION("ndims: 4") {
    test_from_file<TestType>("files/four_dim_large_solve0");
  }
}

TEMPLATE_TEST_CASE("cuda 4D: solveY", "[solvedim:1]", double, float) {
  SECTION("ndims: 4") {
    test_from_file<TestType>("files/four_dim_large_solve1");
  }
}

TEMPLATE_TEST_CASE("cuda 4D: solveZ", "[solvedim:2]", double, float) {
  SECTION("ndims: 4") {
    SECTION("solvedim: 2") {
      test_from_file<TestType>("files/four_dim_large_solve2");
    }
  }
}

TEMPLATE_TEST_CASE("cuda: solve3", "[solvedim:3]", double, float) {
  SECTION("ndims: 4") {
    SECTION("solvedim: 3") {
      test_from_file<TestType>("files/four_dim_large_solve3");
    }
  }
}
#endif
